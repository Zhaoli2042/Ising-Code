#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
//#include <cutil.h>

//INCLUDE THE KERNELS
//#include <neighbor_energy.cu>
#include </home/cwj8781/Ising-Code/CUDA Ising/random_table.cu>
#include </home/cwj8781/Ising-Code/CUDA Ising/Fast_MC.cu>
#include </home/cwj8781/Ising-Code/CUDA Ising/magnet.cu>
#include </home/cwj8781/Ising-Code/CUDA Ising/Parallel_MC.cu>
#include </home/cwj8781/Ising-Code/CUDA Ising/Parallel_MC.h>
#include <stdio.h>

int main()
/* because static memory doesn't support variable size of arrays, 
(F*CK THIS COMPILER) here we let size to be a number, 
remember to change the number, if you want to change the size of the lattice,
also change count*/
{
	int count = 16;
	int8_t *lattice_host;
	static int chose[5][2];
	float T;
//	float m; //Average magnet
	float total_m;
	float total_m2;
/* TOTAL NUMBER OF MONTE CARLO STEPS*/
	int STEP = 1000000;	
	T = 3.5;
	printf("\nT: %f \n", T);
	memset( chose, 1, sizeof(chose) );
	lattice_host = (int8_t*) malloc(count * count * sizeof(int));
	random_table(count, lattice_host);
	printf("\nNEW GENERATED LATTICE IS \n");
	for (int i = 0; i < count; i++){
      	for (int j = 0; j < count; j++){
      	printf("%d\t", lattice_host[i * count + j]);
      	}
      	printf("\n");
	}
	
/* Do memory allocation */
	int steps = 1000;
	int x = count;
	int y = count;
	int z =1 ;
	int n = x* y * z;
	int* total_host = (int*) malloc(sizeof(int*));	
	int val = 0;
	*total_host = val;
	int* total_device = AllocateDeviceTotal(total_host);
	int8_t* lattice_device = AllocateDeviceGrid(lattice_host, n);
	copyToDeviceGrid(lattice_device, lattice_host, n);
	copyToDeviceTotal(total_device, total_host);
	MC(lattice_device, x, y, z, steps,total_device );
	CopyFromDeviceGrid(lattice_host, lattice_device, n);
	CopyFromDeviceTotal(total_host, total_device);
	int t = *total_host;
	printf("%d \n", t);

printf("\nEND OF THE RUNS\n");
//CALCULATE AVERAGE MAGNET
//float mean_m = total_m/STEP;
//float mean_m2 = total_m2/STEP;
//float mean2_m = mean_m * mean_m;

//float ST_D = sqrt(mean_m2 - mean2_m); //STANDARD DEVIATION OF AVERAGE MAGNET
//	printf("\nTOTAL m is : %f\n", total_m);
//	printf("\nMEAN VALUE OF m IS: %f\n", mean_m);
//	printf("\nSTANDARD DEVIATION OF m IS: %f\n", ST_D);
/* TREAT THE DATA, CALCULATE AVERAGES AND STANDARD DEVIATIONS*/

printf("\nFINAL lattice is :\n");
for (int i = 0; i < count; i++){
      for (int j = 0; j < count; j++){
      printf("%d\t", lattice_host[i * count + j]);
      }
      printf("\n");
}

	
	return 0;
}












