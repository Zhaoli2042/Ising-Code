#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <string.h>

#include <iostream>
#include <cstddef>
#define __STDC_FORMAT_MACROS 1
#include <inttypes.h>
#include <cstdio>


#include <math.h>
//#include <cutil.h>
//#include "util.h"
#include "Parallel_MC.h"

__global__ void MC_kernel(int8_t* grid_device,int steps)
{
	//printf("%d \n", grid_device[threadIdx.x]);
//	int tid = threadIdx.x;
//	int bid = blockIdx.x; 	
 
}

__global__ void get_total_kernel(int8_t* grid_device, int* total) 
{
	*total = 20;
	//printf("%d \n", *total);
}



void MC(int8_t* grid_device, size_t x, size_t y, size_t z,  int steps, int* total_device)
{
    /* This function should only contain a call to the GPU 
       histogramming kernel. Any memory allocations and
       transfers must be done outside this function */
     dim3 dimGrid( ceil(x*y*z / 256),1);
     dim3 dimBlock(256,1);
     printf("hey \n");
     MC_kernel<<<dimGrid, dimBlock>>>(grid_device, steps);
	 hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
                printf("error is %s \n",hipGetErrorString(err));
        }
     get_total_kernel<<<dimGrid, dimBlock>>>(grid_device, total_device);	 
}


/* Include below the implementation of any other functions you need */
void copyToDeviceInput(int8_t* grid_device,int8_t* grid_host, unsigned int HEIGHT, unsigned int WIDTH )
{
//printf("copy to device input \n");
size_t size = HEIGHT * WIDTH  * sizeof(int8_t);
  	
hipError_t err = hipMemcpy(grid_device,grid_host,size, hipMemcpyHostToDevice);

if (err != hipSuccess)
{
	printf("error Copy To: %s \n", hipGetErrorString(err));
}
}


void copyToDeviceGrid(int8_t* grid_device, int8_t* grid_host, unsigned int n )
{
	int size = n *  sizeof(int8_t);
        hipError_t err = hipMemcpy(grid_device, grid_host, size, hipMemcpyHostToDevice );
	if (err != hipSuccess)
{
        printf("error in CopyToDeviceGrid: %s \n", hipGetErrorString(err));
}

}

void copyToDeviceTotal(int* total_device, int* total_host)
{
hipError_t err = hipMemcpy(total_device, &total_host, sizeof(int), hipMemcpyHostToDevice );

	if (err != hipSuccess)
{
        printf("error CopyToDevieTotal: %s \n", hipGetErrorString(err));
}

}


void CopyFromDeviceGrid(int8_t* grid_host, int8_t* grid_device, unsigned int n)
{
    int size = n * sizeof(int8_t); 
    hipError_t status = hipMemcpy(grid_host, grid_device, size, hipMemcpyDeviceToHost);
	if (status != hipSuccess) {
	printf("error in CopyFromDeviceGrid:  %s \n", hipGetErrorString(status));
	}
}

void CopyFromDeviceTotal(int* total_host, int* total_device)
{
        hipError_t status = hipMemcpy(total_host, total_device, sizeof(int), hipMemcpyDeviceToHost);
        if (status != hipSuccess) {
        printf("error in CopyFromDeviceTotal:  %s \n", hipGetErrorString(status));
        }

}


void FreeDeviceGrid(int8_t * grid_device)
{
    hipFree(grid_device);
    grid_device = NULL;
}

void FreeDeviceTotal(int* total_device)
{
	hipFree(total_device);
	total_device = NULL;
}

int8_t * AllocateDeviceGrid(int8_t * grid_host, int n)
{
    int8_t * grid_device =  grid_host;
    int size = n * sizeof(uint8_t);
    hipError_t err = hipMalloc((void**) &grid_device, size);
	if (err != hipSuccess)
{
        printf("error Allocate device grid: %s \n", hipGetErrorString(err));
}

    return grid_device;
}

int* AllocateDeviceTotal(int* total_host)
{
	int* total_device = total_host;
	int size =  sizeof(int);
	hipError_t err = hipMalloc((void**) &total_device, size);
	if (err != hipSuccess)
{
        printf("error Copy To: %s \n", hipGetErrorString(err));
}

	return total_device;
}

int8_t * convert(int8_t** input, int HEIGHT, int WIDTH)
{
	int8_t* one_d = (int8_t*)malloc(sizeof(int8_t) * WIDTH * HEIGHT);
	for (int i =0 ; i < HEIGHT; i++)
	{
	for (int j= 0; j<WIDTH; j++)
	{
	     one_d[i * WIDTH + j] = input[i][j];
	}
	}

	return one_d;

}
