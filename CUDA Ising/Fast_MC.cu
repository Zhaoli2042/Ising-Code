#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include </home/zlc6394/project/r2.cu>
#include </home/zlc6394/project/neighbor_energy.cu>


void Fast_MC(int count, int* lattice, int* new_lattice, double T, int chose_x, int chose_y)
{
	
// FLIP THE CHOSEN ONE!
	new_lattice[chose_x * count + chose_y] = (-1) * lattice[chose_x * count + chose_y];

/* CALCULATE 1st NEIGHBORS*/
		
		int chose[5][2] = {
		{chose_x, chose_y+1},
		{chose_x, chose_y-1},
		{chose_x-1, chose_y},
		{chose_x+1, chose_y},
		{chose_x, chose_y}
		};
	for (int i = 0; i < 5; i++){
/*		printf("\nNeighbor and self(last): (%d, %d)\n", chose[i][0], chose[i][1]);*/
	}
/*	printf("\n");*/
	
	if ((chose_y+1) > (count-1)) {
		chose[0][0] = chose_x;
		chose[0][1] = 0;
	}
	
	if ((chose_y-1) < 0) {
		chose[1][0] = chose_x;
		chose[1][1] = count - 1;
	} 

	if ((chose_x-1) < 0) {
		chose[2][0] = count - 1;
		chose[2][1] = chose_y;
	} 
	
	if ((chose_x+1) > (count-1)) {
		chose[3][0] = 0;
		chose[3][1] = chose_y;
	} 
/* 	for (int i = 0; i < 5; i++){
		printf("\n Neighbor and self(last): (%d, %d)\n", chose[i][0], chose[i][1]);
	} */
	

/* Then we need to call neigh_energy and plug in the old lattice to calculate the old energy*/
/* COMPUTE THE OLD CONFIGURATION ENERGY*/
	int old_spin;
	old_spin = 0;
	int e;
	double old_energy;
	for (int l = 0; l < 5; l++){

		e = neighbor_energy(count, chose[l][0], chose[l][1], lattice);
// 		printf("\n old energy of that site is: %d\n", e);  
		old_spin += e * lattice[chose[l][0] * count + chose[l][1]];
		/*NOTE THAT HERE WE USED SPIN AT SITE TIMES SUM OF SURROUNDING SPINS*/

	}
// 	printf("\nTotal Spin of old configuration is: %d\n", old_spin);  
	old_energy = old_spin * (-1) * (0.5);
// 	printf("OLD CONFIGURATION ENERGY IS: %f", old_energy);  
/* COMPUTE THE ENERGY OF THE NEW CONFIGURATION*/
	int new_spin;
	new_spin = 0;
	double new_energy;
	for (int l = 0; l < 5; l++){

		e = neighbor_energy(count, chose[l][0], chose[l][1], new_lattice);
// 		printf("\n new energy of that site is: %d\n", e);  
		new_spin += e * new_lattice[chose[l][0] * count + chose[l][1]];
		/*NOTE THAT HERE WE USED SPIN AT SITE TIMES SUM OF SURROUNDING SPINS*/
	}
// 	printf("\nTotal Spin of new configuration is: %d\n", new_spin);
 	new_energy = new_spin * (-1) * (0.5);
// 	printf("NEW CONFIGURATION ENERGY IS: %f", new_energy); 
/* TO CALCULATE DELTA IN ENERGY, DELTA_ENERGY = NEW - OLD */
	double delta_energy;
	delta_energy = new_energy - old_energy;
//	printf("DELTA_ENERGY is: %f", delta_energy);
	double acc_prob;
	double boltz_factor;
	boltz_factor = exp(- delta_energy / T);
// 	printf("\nBOLTZMANN FACTOR IS: %f", boltz_factor); 
	acc_prob = min((double) 1, boltz_factor);
// 	printf("\n ACCEPTIING PROBABILITY IS: %f", acc_prob); 
	double random_num;
	random_num = r2();
// 	printf("\nRANDOM NUMBER IS: %f", random_num);
	if (random_num < acc_prob){
// 		printf("\nACCEPT\n"); 
// 		printf("OUR NEW LATTICE IS: \n"); 
			for (int i = 0; i < count; i++){
				for (int j = 0; j < count; j++){
				lattice[i * count + j] = new_lattice[i * count + j];
// 				printf("%d\t", new_lattice[i * count + j]); 
				}
//				printf("\n");
				}
	}
}

//double min(double a, double b){
//	double out;
//	out = b;
//	if (a < b){
//		out = a;
//	}
//	return out;
//}
