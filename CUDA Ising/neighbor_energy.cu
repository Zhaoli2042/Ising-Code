
#include <hip/hip_runtime.h>
int neighbor_energy (int count, int i, int j, int* lattice)
{
/* IN 2D SYSTEM, THERE ARE 4 NEAREST NEIGHBORS*/
	int neigh[4][2] = {
		{i, j+1},
		{i, j-1},
		{i-1, j},
		{i+1, j}
	};
	if ((j+1) > (count-1)) {
		neigh[0][0] = i;
		neigh[0][1] = 0;
	} 
	
	if ((j-1) < 0) {
		neigh[1][0] = i;
		neigh[1][1] = count - 1;
	} 

	if ((i-1) < 0) {
		neigh[2][0] = count - 1;
		neigh[2][1] = j;
	} 
	
	if ((i+1) > (count-1)) {
		neigh[3][0] = 0;
		neigh[3][1] = j;
	} 
/*	printf("Site is %d, %d: \t", i, j); */
	int k;
	int sum_neigh = 0;
/* 	printf("\nWE ARE AT SITE (%d, %d)\n", i, j); */
	for (k = 0; k < 4; k++)
	{	
/* 		printf("\n neigh is: (%d, %d)", neigh[k][0], neigh[k][1]);
		printf(" Spin of that neighbor: %d", lattice[neigh[k][0]][neigh[k][1]]);
 */		sum_neigh += lattice[neigh[k][0] * count + neigh[k][1]];
		
	}
/* 	printf("\nSum of neighboring spins is: %d\n", sum_neigh);
	
	printf("\n\n\n\n\n"); */
/* PRINT OUT ENERGY OF THIS SINGLE SITE*/
	return sum_neigh;
}
