
#include <hip/hip_runtime.h>

/*CALCULATE THE MAGNETIZATION PER SPIN OF THIS LATTICE */
float magnet_per_spin(int count, int* lattice){
	float total_site = (float) count * count;
	float TOTAL_SPIN = 0;
	float m;;
	for (int i = 0; i < count; i++){
		for (int j = 0; j < count; j++){
			TOTAL_SPIN += lattice[i * count + j];
		}
	}
	m = TOTAL_SPIN/total_site;
 	//printf("\nTOTAL SPIN IS: %f\n", TOTAL_SPIN);
	//printf("\nMAG PER SPIN IS: %f\n", m); 
	
	return m;
}
