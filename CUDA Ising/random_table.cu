#include "hip/hip_runtime.h"
/* CREATES A RANDOM ISING SPIN LATTICE */
#ifndef _random_H_
#define _random_H_

#include <stdio.h>
#include "Ising.h"
void random_table(int count, int* lattice)
{
	int i;
	int j;
	printf("\nHERE WE PRINT OUT OUR INITIALIZED MATRIX\n");
	for (i = 0; i < count; i++)
	{
		for (j = 0; j < count; j++)
		{
			lattice[i * count + j] = (int) pow(-1,rand());
		}
		printf("\n");
	}
		printf("\nOUR INITIAL LATTICE IS\n");
	for (i = 0; i < count; i++)
    {
        for (j = 0; j < count; j++)
        {
            printf("%d\t", lattice[i * count + j]);
        }
        printf("\n");
    }
}

#endif // #ifndef _random_H_
