#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
//#include <cutil.h>

//INCLUDE THE KERNELS
//#include <neighbor_energy.cu>
#include </home/zlc6394/project/random_table.cu>
#include </home/zlc6394/project/Fast_MC.cu>
#include </home/zlc6394/project/magnet.cu>
#include <stdio.h>

int main()
/* because static memory doesn't support variable size of arrays, 
(F*CK THIS COMPILER) here we let size to be a number, 
remember to change the number, if you want to change the size of the lattice,
also change count*/
{
	int count = 16;
	int *lattice;
	static int chose[5][2];
	int *new_lattice;
	
	float T;
//	float m; //Average magnet
	float total_m;
	float total_m2;
/* TOTAL NUMBER OF MONTE CARLO STEPS*/
	int STEP = 1000000;	
	T = 3.5;
	printf("\nT: %f \n", T);
	memset( chose, 1, sizeof(chose) );
	lattice = (int*) malloc(count * count * sizeof(int));
	new_lattice = (int*) malloc(count * count * sizeof(int));
	random_table(count, lattice);
	printf("\nNEW GENERATED LATTICE IS \n");
	for (int i = 0; i < count; i++){
      	for (int j = 0; j < count; j++){
      	printf("%d\t", lattice[i * count + j]);
      	}
      	printf("\n");
	}

	
	
	
	int chose_x;
	int chose_y;		
/* RUN THROUGH THESE MC STEPS */
for (int t = 0; t < STEP; t++){

chose_x = rand()%(count);
chose_y = rand()%(count);
// COPY ORIGINAL LATTICE TO THE NEW LATTICE
//printf("\nFlipped element is [%d, %d]\n", chose_x, chose_y);
//printf("\nCOPIED and CHANGED NEW LATTICE IS: \n");
//memcpy(new_lattice, lattice, count*count);
        for (int i = 0; i < count; i++)
    {
        for (int j = 0; j < count; j++)
        {
        //printf("\nHERE j is %d\n", j);
	new_lattice[i * count + j] = lattice[i * count + j];
//         printf("%i\t", new_lattice[i * count + j]);
                }
//        printf("\n");
    }

Fast_MC(count, lattice, new_lattice, T, chose_x, chose_y);
//printf("\nflipped new lattice is :\n");
//for (int i = 0; i < count; i++){
//	for (int j = 0; j < count; j++){
//	printf("%d\t", lattice[i * count + j]);
//	}
//	printf("\n");
//}
//CALCULATE TOTAL MAGNET OF THIS RUN
float m = magnet_per_spin(count, lattice);
//printf("step is %d, m is %f", STEP, m);
total_m += m;
total_m2 += m*m;
}

printf("\nEND OF THE RUNS\n");
//CALCULATE AVERAGE MAGNET
float mean_m = total_m/STEP;
float mean_m2 = total_m2/STEP;
float mean2_m = mean_m * mean_m;

float ST_D = sqrt(mean_m2 - mean2_m); //STANDARD DEVIATION OF AVERAGE MAGNET
	printf("\nTOTAL m is : %f\n", total_m);
	printf("\nMEAN VALUE OF m IS: %f\n", mean_m);
	printf("\nSTANDARD DEVIATION OF m IS: %f\n", ST_D);
/* TREAT THE DATA, CALCULATE AVERAGES AND STANDARD DEVIATIONS*/

printf("\nFINAL lattice is :\n");
for (int i = 0; i < count; i++){
      for (int j = 0; j < count; j++){
      printf("%d\t", lattice[i * count + j]);
      }
      printf("\n");
}

	
	return 0;
}












